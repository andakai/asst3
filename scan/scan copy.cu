#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}


__global__ void upsweep(int twod, int N, int *result) {
    int twod1 = 2 * twod;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // tid * twod1为tid号线程对应的处理的位置
    // for循环是防止线程数少，而N过大
    // for (int i = tid; i * twod1 < N; i += gridDim.x * blockDim.x) {
    //     int idx = i * twod1;
    //     result[idx + twod1 - 1] += result[idx + twod - 1];
    // }
    int i = tid * twod1;
    if (i < N) {
        result[i + twod1 - 1] += result[i + twod - 1];
    }

}

__global__ void downsweep(int twod, int N, int *result) {
    int twod1 = 2 * twod;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // for (int i = tid; i * twod1 < N; i += gridDim.x * blockDim.x) {
    //     int idx = i * twod1;
    //     int t = result[idx + twod - 1];
    //     result[idx + twod - 1] = result[idx + twod1 - 1];
    //     result[idx + twod1 - 1] += t;
    // }
    int i = tid * twod1;
    if (i < N) {
        int t = result[i + twod - 1];
        result[i + twod - 1] = result[i + twod1 - 1];
        result[i + twod1 - 1] += t;
    }
}

__global__ void set_element_to_zero(int *array, int N) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        array[N - 1] = 0;
    }
}
// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    N = nextPow2(N);
    // upsweep
    for (int twod = 1; twod <= N / 2; twod *= 2) {
        int twod1 = twod * 2;
        // 总共需要的线程数
        int totalThreadsNum = (N + twod1 - 1) / twod1;
        int blockNum = (totalThreadsNum + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        // 在调用exclusive_scan时传入的result就和input的结果相同了，所以可以直接在result上改变。
        // !!!如果所需要的线程不足一个blocknum时，不能使用一个block的所有线程，不然的话，其他位置的元素会干扰本轮结果
        if (totalThreadsNum < THREADS_PER_BLOCK) {
            upsweep<<<1, totalThreadsNum>>>(twod, N, result);
        }
        else {
            upsweep<<<blockNum, THREADS_PER_BLOCK>>>(twod, N, result);
        }
        hipDeviceSynchronize();
    }

    set_element_to_zero<<<1, THREADS_PER_BLOCK>>>(result, N);
    hipDeviceSynchronize();

    // downsweep
    for (int twod = N / 2; twod >= 1; twod /= 2) {
        int twod1 = twod * 2;
        int totalThreadsNum = (N + twod1 - 1) / twod1;
        int blockNum = (totalThreadsNum + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        if (totalThreadsNum < THREADS_PER_BLOCK) {
            downsweep<<<1, totalThreadsNum>>>(twod, N, result);
        }
        else {
            downsweep<<<blockNum, THREADS_PER_BLOCK>>>(twod, N, result);
        }
        
        hipDeviceSynchronize();
    }
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


__global__ void flagRepeat(int* input, int length, int *flags, int* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < length - 1; i += stride) {
        if (input[i] == input[i + 1])
            output[i] = flags[i] = 1;
        else
            output[i] = flags[i] = 0;
    }
}

__global__ void record_indices(int* flags, int* prefix_sum, int length, int* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < length - 1; i += stride) {
        if (flags[i] == 1) {
            int pos = prefix_sum[i];
            output[pos] = i;
        }
    }
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
   
    int *device_flags;
    int *prefix_sum;
    hipMalloc(&device_flags, length * sizeof(int));
    hipMalloc(&prefix_sum, length * sizeof(int));
    //每个线程处理一个数
    int blockNum = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    flagRepeat<<<blockNum, THREADS_PER_BLOCK>>>(device_input, length, device_flags, prefix_sum);
    
    hipDeviceSynchronize();

    // exclusive scan
    exclusive_scan(device_input, length, prefix_sum);


    //exclusive前缀和的最后一个元素为重复元素总个数。
    int total_repeats;
    hipMemcpy(&total_repeats, prefix_sum + length - 1, sizeof(int), hipMemcpyDeviceToHost);
    
    //得到重复元素下标的数组
    record_indices<<<blockNum, THREADS_PER_BLOCK>>>(device_flags, prefix_sum, length, device_output);
    hipDeviceSynchronize();


    hipFree(device_flags);

    return total_repeats; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
